#include <stdio.h>
#include <time.h>
#include <hipblas.h>
#include <hipblas.h>

#define indexCalculation(i,j,ld) (((j)*(ld))+(i))

/**
* printMatrix: A method that takes in a matrix and its dimentions and it prints
*/
void printMatrix(const char* name, float *matrix, int matrixWidth, int matrixHeight) {
	printf("\n%s: [ ", name);
	for(int i = 0 ; i < matrixHeight ; i++) {
		printf("\n");
		for(int j = 0 ; j < matrixWidth ; j++) {
			printf("%f ,", matrix[indexCalculation(i, j, matrixHeight)]);
		}
	}
	printf(" ]", name);
}

/**
* runOperation: Taking the number of blocks and threads it does an operation on the two 
* given matrices and prints their results.
*/
void runOperation(int matrixHeight, int matrixWidth) { 
    
	// Setup Timing Variables
	hipEvent_t start, stop; 
	float elapsedTimeInMiliseconds; 
	hipEventCreate(&start); 
	hipEventCreate(&stop); 
  
	// Setup CUDA Streams
	hipStream_t operationStream; 
	hipStreamCreate(&operationStream); 

	// Setup host memory variables
    cublasInit();
	hipblasHandle_t handle;
	hipblasCreate(&handle);

    float *mA = (float*) malloc(matrixHeight * matrixWidth * sizeof(float));
    float *mB = (float*) malloc(matrixHeight * matrixWidth * sizeof(float));
    float *mC = (float*) malloc(matrixHeight * matrixWidth * sizeof(float));

    for (int i = 0 ; i < matrixHeight ; i++) {
      	for (int j = 0 ; j < matrixWidth ; j++) {
        	mA[indexCalculation(i,j,matrixHeight)] = (float) indexCalculation(i,j,matrixHeight);
			mB[indexCalculation(i,j,matrixHeight)] = (float) indexCalculation(i,j,matrixHeight); 
		}   
	}
    
	// Turned off to minimize printing
	printMatrix("Matrix A", mA, matrixWidth, matrixHeight);
	printMatrix("Matrix B", mB, matrixWidth, matrixHeight);

	// Start Stream
	hipblasSetStream(handle, operationStream);

	// Setup device memory variables
	float* dev_mA; float* dev_mB; float* dev_mC;
	cublasAlloc(matrixHeight * matrixWidth, sizeof(float), (void**) &dev_mA);
	cublasAlloc(matrixHeight * matrixWidth, sizeof(float), (void**) &dev_mB);
	cublasAlloc(matrixHeight * matrixWidth, sizeof(float), (void**) &dev_mC);

	// Copy matrix from host to device memory
	hipblasSetMatrix(matrixHeight, matrixWidth, sizeof(float), mA, matrixHeight, dev_mA, matrixHeight);
	hipblasSetMatrix(matrixHeight, matrixWidth, sizeof(float), mB, matrixHeight, dev_mB, matrixHeight);

    // Execute Multiplication
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matrixHeight, matrixWidth, matrixWidth, alpha, dev_mA, matrixHeight, dev_mB, matrixHeight, beta, dev_mC, matrixHeight);

	// Get result
    hipblasGetMatrix(matrixHeight, matrixWidth, sizeof(float), dev_mC, matrixHeight, mC, matrixHeight);

    // Timing Output
	hipStreamSynchronize(operationStream);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop); 
	hipEventElapsedTime(&elapsedTimeInMiliseconds, start, stop); 
  	printf("Stream and Event Time: %f Miliseconds\n", elapsedTimeInMiliseconds) * 100;

  	// Turned off to minimize printing
	printMatrix("Matrix C", mC, matrixWidth, matrixHeight);

	// Free reserved memory
    free(mA); 
	free(mB);
	free(mC);
	cublasFree(dev_mA);
	cublasFree(dev_mB);
	cublasFree(dev_mC);
	hipblasDestroy(handle);
	cublasShutdown();
}

int main(int argc, char** argv) {
	int matrixHeight = 1024;

	// read command line arguments
	if (argc >= 2) {
		matrixHeight = atoi(argv[1]);
	}

	printf("\nMatrix Width and Height: %d\n", matrixHeight);
	
	runOperation(matrixHeight, matrixHeight);

	return 0;
}
