#include <stdio.h>
#include <time.h>
#define indexCalculation(i,j,ld) (((j)*(ld))+(i))

/**
* printMatrix: A method that takes in a matrix and its dimentions and it prints
*/
void printMatrix(const char* name, float *matrix, int matrixWidth, int matrixHeight) {
	printf("\n%s: [ ", name);
	for(int i = 0 ; i < matrixHeight ; i++) {
		printf("\n");
		for(int j = 0 ; j < matrixWidth ; j++) {
			printf("%f ,", matrix[indexCalculation(i, j, matrixHeight)]);
		}
	}
	printf(" ]", name);
}

/**
* runOperation: Taking the number of blocks and threads it does an operation on the two 
* given matrices and prints their results.
*/
void runOperation(int matrixHeight, int matrixWidth) { 
    
	// Setup Timing Variables
	hipEvent_t start, stop; 
	float elapsedTimeInMiliseconds; 
	hipEventCreate(&start); 
	hipEventCreate(&stop); 
  
	// Setup CUDA Streams
	hipStream_t operationStream; 
	hipStreamCreate(&operationStream); 
  
	// Setup host memory variables
    cublasInit();

    float *mA = (float*) malloc(matrixHeight * matrixWidth * sizeof(float));
    float *mB = (float*) malloc(matrixHeight * matrixWidth * sizeof(float));
    float *mC = (float*) malloc(matrixHeight * matrixWidth * sizeof(float));

    for (int i = 0 ; i < matrixHeight ; i++) {
      	for (int j = 0 ; j < matrixWidth ; j++) {
        	A[index(i,j,matrixHeight)] = (float) index(i,j,matrixHeight);
			B[index(i,j,matrixHeight)] = (float) index(i,j,matrixHeight); 
		}   
	}
    
	// Turned off to minimize printing
	printMatrix("Matrix A", mA, matrixWidth, matrixHeight);
	printMatrix("Matrix B", mB, matrixWidth, matrixHeight);

	// Start Stream
	hipblasSetStream(operationStream);

	// Setup device memory variables
	float* dev_mA; float* dev_mB; float* dev_mC;
	cublasAlloc(matrixHeight * matrixWidth, sizeof(float), (void**) &dev_mA);
	cublasAlloc(matrixHeight * matrixWidth, sizeof(float), (void**) &dev_mB);
	cublasAlloc(matrixHeight * matrixWidth, sizeof(float), (void**) &dev_mC);

	// Copy matrix from host to device memory
	hipblasSetMatrix(matrixHeight, matrixWidth, sizeof(float), mA, matrixHeight, dev_mA, matrixHeight);
	hipblasSetMatrix(matrixHeight, matrixWidth, sizeof(float), mB, matrixHeight, dev_mB, matrixHeight);

    // Execute Multiplication
    hipblasSgemm('n','n', matrixHeight, matrixWidth, matrixWidth,1 /* Alpha */, dev_mA, matrixHeight, dev_mB, matrixHeight, 0 /* Beta */, dev_mC, matrixHeight);

	// Get result
    hipblasGetMatrix(matrixHeight, matrixWidth, sizeof(float), dev_mC, matrixHeight, mC, matrixHeight);

    // Timing Output
	hipStreamSynchronize(operationStream);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop); 
	hipEventElapsedTime(&elapsedTimeInMiliseconds, start, stop); 
  	printf("Stream and Event Time: %f Miliseconds\n", elapsedTimeInMiliseconds) * 100;

  	// Turned off to minimize printing
	printMatrix("Matrix C", mC, matrixWidth, matrixHeight);

	// Free reserved memory
    free(mA); 
	free(mB);
	free(mC);
	cublasFree(dev_mA);
	cublasFree(dev_mB);
	cublasFree(dev_mC);
}

int main(int argc, char** argv) {
	int matrixHeight = 1024;

	// read command line arguments
	if (argc >= 2) {
		matrixHeight = atoi(argv[1]);
	}

	printf("\nMatrix Width and Height: %d\n", matrixHeight);
	
	runOperations(matrixHeight, matrixHeight);

	return 0;
}
