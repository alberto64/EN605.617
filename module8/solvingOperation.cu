#include <stdio.h>
#include <hipblas.h>
#include <hipblas.h>
#include <hipsolver.h>

#define indexCalculation(i,j,ld) (((j)*(ld))+(i))

/**
* printMatrix: A method that takes in a matrix and its dimentions and it prints
*/
void printMatrix(const char* name, double *matrix, int matrixWidth, int matrixHeight) {
	printf("\n%s: [ ", name);
	for(int i = 0 ; i < matrixHeight ; i++) {
		printf("\n");
		for(int j = 0 ; j < matrixWidth ; j++) {
			printf("%f, ", matrix[indexCalculation(i, j, matrixHeight)]);
		}
	}
	printf("]");
}

/**
* runOperation: Taking the number of blocks and threads it does an operation on the two 
* given matrices and prints their results.
*/
void runOperation(int matrixHeight, int matrixWidth, int nrhs) { 
    
	// Setup Timing Variables
	hipEvent_t start, stop; 
	float elapsedTimeInMiliseconds; 
	hipEventCreate(&start); 
	hipEventCreate(&stop); 

	// Setup host memory variables
    cublasInit();
	
	// Setup Handle and stream
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipsolverHandle_t solver;
	hipsolverDnCreate(&solver);
	hipStream_t operationStream; 
	hipStreamCreate(&operationStream);
	hipblasSetStream(handle, operationStream);

    double *mA = (double*) malloc(matrixHeight * matrixWidth * sizeof(double));
    double *vB = (double*) malloc(matrixHeight * nrhs * sizeof(double));
    double *mX = (double*) malloc(matrixHeight * nrhs * sizeof(double));

    for (int i = 0 ; i < matrixHeight ; i++) {
      	for (int j = 0 ; j < matrixWidth ; j++) {
        	mA[indexCalculation(i,j,matrixHeight)] = (double) indexCalculation(i,j,matrixHeight);
		}   
		for (int j = 0 ; j < nrhs; j++) {
			vB[indexCalculation(i,j,matrixHeight)] = (double) indexCalculation(i,j,matrixHeight); 
		}   

	}
    
	// Turned off to minimize printing
	printMatrix("Matrix A", mA, matrixWidth, matrixHeight);
	printMatrix("Vector B", vB, nrhs, matrixHeight);

	// Setup device memory variables
	int* dev_Info; 
	int  lwork = 0; 
	double* dev_mA; double* dev_vB; double* dev_tau; double *dev_work;
	const double one = 1;

	hipMalloc((void**) &dev_mA  , sizeof(double) * matrixHeight * matrixWidth);
    hipMalloc((void**) &dev_tau, sizeof(double) * matrixHeight);
    hipMalloc((void**) &dev_vB  , sizeof(double) * matrixHeight * nrhs);
    hipMalloc((void**) &dev_Info, sizeof(int));

	hipEventRecord(start);

    hipMemcpy(dev_mA, mA, sizeof(double) * matrixHeight * matrixWidth, hipMemcpyHostToDevice);
    hipMemcpy(dev_vB, vB, sizeof(double) * matrixHeight * nrhs, hipMemcpyHostToDevice);
 
	// Query working space of geqrf and ormqr
    hipsolverDnDgeqrf_bufferSize(solver, matrixHeight, matrixWidth, dev_mA, matrixHeight, &lwork);
	hipMalloc((void**) &dev_Info, sizeof(double) * lwork);

	// Compute QR factorization
	hipsolverDnDgeqrf(solver, matrixHeight, matrixWidth, dev_mA, matrixHeight, dev_tau, dev_work, lwork, dev_Info);
    hipDeviceSynchronize();

	// Compute Q^T*B
    hipsolverDnDormqr(solver, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, matrixHeight, nrhs, matrixWidth, dev_mA, matrixHeight,
        dev_tau, dev_vB, matrixHeight, dev_work, lwork, dev_Info);
    hipDeviceSynchronize();

	// Compute x = R \ Q^T*B

    hipblasDtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, matrixHeight,
        nrhs, &one, dev_mA, matrixHeight, dev_vB, matrixHeight);
    hipDeviceSynchronize();

    hipMemcpy(mX, dev_vB, sizeof(double) * matrixHeight * nrhs, hipMemcpyDeviceToHost);

    // Timing Output
	hipStreamSynchronize(operationStream);
	hipblasDestroy(handle);
	hipsolverDnDestroy(solver);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop); 
	hipEventElapsedTime(&elapsedTimeInMiliseconds, start, stop); 

  	// Turned off to minimize printing
	printMatrix("Result X", mX, nrhs, matrixHeight);
	printf("\nStream and Event Time: %f Miliseconds\n", elapsedTimeInMiliseconds) * 100;

	// Free reserved memory
    free(mA); 
	free(vB);
	free(mX);
	hipFree(dev_mA);
	hipFree(dev_vB);
	hipFree(dev_tau);
	hipFree(dev_work);
	hipFree(dev_Info);
	cublasShutdown();
	hipDeviceReset();
}

int main(int argc, char** argv) {
	int matrixHeight = 1024;

	// read command line arguments
	if (argc >= 2) {
		matrixHeight = atoi(argv[1]);
	}

	printf("\nMatrix Width and Height: %d\n", matrixHeight);
	
	runOperation(matrixHeight, matrixHeight, 1);

	return 0;
}
