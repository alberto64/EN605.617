
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

// Initialize constant variables
__constant__ int* constThreadCountList;
__constant__ int* constRandNumList;
__constant__ int* constAddresultList;

/**
* addConstCuda: A method that add two arrays and places the result in a third array using 
* multithreading for index calculation using constant memory.
*/
__global__ void addConstCUDA(int* resultList) { 
	const int idx = threadIdx.x + (blockIdx.x * blockDim.x); 
	resultList[idx] = constThreadCountList[idx] + constRandNumList[idx]; 
}

/**
* subConstCuda: A method that substract two arrays and places the result in a third array using 
* multithreading for index calculation using constant memory.
*/
__global__ void subConstCUDA(int* resultList) { 
	const int idx = threadIdx.x + (blockIdx.x * blockDim.x); 
	resultList[idx] = constThreadCountList[idx] - constRandNumList[idx]; 
}

/**
* multConstCuda: A method that multiplies two arrays and places the result in a third array using 
* multithreading for index calculation using constant memory.
*/
__global__ void multConstCUDA(int* resultList) { 
	const int idx = threadIdx.x + (blockIdx.x * blockDim.x); 
	resultList[idx] = constThreadCountList[idx] * constRandNumList[idx]; 
}

/**
* modConstCuda: A method that does the modulus between two arrays and places the result in a third 
* array using multithreading for index calculation using constant memory.
*/
__global__ void modConstCUDA(int* resultList) { 
	const int idx = threadIdx.x + (blockIdx.x * blockDim.x); 
	resultList[idx] = constThreadCountList[idx] % constRandNumList[idx]; 
}

/**
* printArray: A method that takes in an a label and an array with its size and it feeds it to printf.
*/
void printArray(const char* name, const int *array, const int size) {
	printf("\n%s: [ ", name);
	for(int idx = 0; idx < size; idx++) {
		printf("%i ", array[idx]);
	}
	printf("]\n");
}

/**
* runOperations: Taking the number of blocks and threads it does 4 operations on the two 
* given arrays and prints their results. Uses const memory
*/
void runOperations(int numBlocks, int totalThreads, int *threadCountList, int *randNumList) { 
	

	// Set up input constant variables
	hipMemcpyToSymbol(HIP_SYMBOL(constThreadCountList), &threadCountList, sizeof(int) * totalThreads);
	hipMemcpyToSymbol(HIP_SYMBOL(constRandNumList), &randNumList, sizeof(int) * totalThreads);

	// Prepare result array variables
	int* addresultList = (int*) malloc(totalThreads * sizeof(int));
	int* subresultList = (int*) malloc(totalThreads * sizeof(int));
	int* multresultList = (int*) malloc(totalThreads * sizeof(int));
	int* modresultList = (int*) malloc(totalThreads * sizeof(int));
    int *dev_result;

	hipMalloc((void **)&dev_result, totalThreads * sizeof(int));

	// Execute each operation and bring result from device to host
	addConstCUDA<<<numBlocks,totalThreads>>> (dev_result);
	hipDeviceSynchronize();
	hipMemcpy(&addresultList, dev_result, sizeof(int) * totalThreads, hipMemcpyDeviceToHost);

	subConstCUDA<<<numBlocks,totalThreads>>> (dev_result);
	hipDeviceSynchronize();
	hipMemcpy(&subresultList, dev_result, sizeof(int) * totalThreads, hipMemcpyDeviceToHost);

	multConstCUDA<<<numBlocks,totalThreads>>> (dev_result);
	hipDeviceSynchronize();
	hipMemcpy(&multresultList, dev_result, sizeof(int) * totalThreads, hipMemcpyDeviceToHost);

	modConstCUDA<<<numBlocks,totalThreads>>> (dev_result);
	hipDeviceSynchronize();
	hipMemcpy(&modresultList, dev_result, sizeof(int) * totalThreads, hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	
	// Turned of to minimize printing
	printArray("Add Result", addresultList, totalThreads);
	printArray("Sub Result", subresultList, totalThreads);
	printArray("Mult Result", multresultList, totalThreads);
	printArray("Mod Result", modresultList, totalThreads);
}

/**
* runTest: Used to set up variables needed to run a timing test.
*/ 
void runTest(const int numBlocks, const int totalThreads) {

	// Set up variables for timing
	clock_t start, end;
	double timePassedMiliSeconds;

	// Set up global memory space 
	int* threadCountList = (int*) malloc(totalThreads * sizeof(int));
	int* randNumList = (int*) malloc(totalThreads * sizeof(int));
	
	// Populate paged memory arrays
	for ( int idx = 0; idx < totalThreads; idx++ ) {
    	threadCountList[idx] = idx; 
		randNumList[idx] = rand() % 4;
   	}
	
	// Show generated values
	// Turned of to minimize printing
	// printArray("Thread Count List", threadCountList, totalThreads);
	// printArray("Random Number List", randNumList, totalThreads);
	
	// Run and time operations using const memory
	start = clock();
	runOperations(numBlocks, totalThreads, threadCountList, randNumList);
	end = clock();
	timePassedMiliSeconds = (double) (end - start) * 1000 / CLOCKS_PER_SEC;
	printf("\nConstant Memory Time: %f Miliseconds\n", timePassedMiliSeconds);

	// Free device memory
	hipDeviceReset();
}

/**
* Main method: starts the execution.
*/
int main(int argc, char** argv) {
	// Based on the work of Andrew Krepps
	
	// Set default values in case arguments don't come in command line.
	int totalThreads = 1024;
	int blockSize = 256;

	// read command line arguments
	if (argc >= 2) {
		totalThreads = atoi(argv[1]);
	}
	if (argc >= 3) {
		blockSize = atoi(argv[2]);
	}

	int numBlocks = totalThreads/blockSize;

	// validate command line arguments
	if (totalThreads % blockSize != 0) {
		++numBlocks;
		totalThreads = numBlocks*blockSize;
		
		printf("Warning: Total thread count is not evenly divisible by the block size\n");
		printf("The total number of threads will be rounded up to %d\n", totalThreads);
	}

	printf("Total Threads: %d\nBlock Size: %d\n", totalThreads, blockSize);

	runTest(numBlocks, totalThreads);

	return 0;
}