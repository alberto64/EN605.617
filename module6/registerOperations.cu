
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

/**
* addCuda: A method that add two arrays and places the result in a first array using 
* multithreading for index calculation.
*/
__global__ void addCUDA(int *threadCountList, const int *randNumList) { 
	const int idx = threadIdx.x + (blockIdx.x * blockDim.x); 
	int operation = threadCountList[idx];
	operation = operation + randNumList[idx]; 
	threadCountList[idx] = operation; 
}

/**
* subCuda: A method that substract two arrays and places the result in a first array using 
* multithreading for index calculation.
*/
__global__ void subCUDA(int *threadCountList, const int *randNumList) { 
	const int idx = threadIdx.x + (blockIdx.x * blockDim.x); 
	int operation = threadCountList[idx];
	operation = operation + randNumList[idx]; 
	threadCountList[idx] = operation; 
}

/**
* multCuda: A method that multiplies two arrays and places the result in a first array using 
* multithreading for index calculation.
*/
__global__ void multCUDA(int *threadCountList, const int *randNumList) { 
	const int idx = threadIdx.x + (blockIdx.x * blockDim.x); 
	int operation = threadCountList[idx];
	operation = operation + randNumList[idx]; 
	threadCountList[idx] = operation; 
}

/**
* modCuda: A method that does the modulus between two arrays and places the result in a first 
* array using multithreading for index calculation.
*/
__global__ void modCUDA(int *threadCountList, const int *randNumList) { 
	const int idx = threadIdx.x + (blockIdx.x * blockDim.x); 
	int operation = threadCountList[idx];
	operation = operation + randNumList[idx]; 
	threadCountList[idx] = operation;  
}

/**
* printArray: A method that takes in an a label and an array with its size and it feeds it to printf.
*/
void printArray(const char* name, int *array, int size) {
	printf("\n%s: [ ", name);
	for(int idx = 0; idx < size; idx++) {
		printf("%i ", array[idx]);
	}
	printf("]\n");
}

/**
* runOperations: Taking the number of blocks and threads it does 4 operations on the two 
* given arrays and prints their results. Uses paged memory
*/
void runOperations(int numBlocks, int totalThreads, int* threadCountList, int* randNumList) { 
	
	// Prepare result array variables
	int* addresultList = (int*) malloc(totalThreads * sizeof(int));
	int* subresultList = (int*) malloc(totalThreads * sizeof(int));
	int* multresultList = (int*) malloc(totalThreads * sizeof(int));
	int* modresultList = (int*) malloc(totalThreads * sizeof(int));
	
	// Prepare cuda variables
	int *dev_addresultList, *dev_subresultList, *dev_multresultList, *dev_modresultList, *dev_randNumList;
	hipMalloc((void**)&dev_addresultList, totalThreads * sizeof(int));
	hipMalloc((void**)&dev_subresultList, totalThreads * sizeof(int));
	hipMalloc((void**)&dev_multresultList, totalThreads * sizeof(int));
	hipMalloc((void**)&dev_modresultList, totalThreads * sizeof(int));
	hipMalloc((void**)&dev_randNumList, totalThreads * sizeof(int));

	// Copy inputs into device memory 
	hipMemcpy(dev_addresultList, threadCountList, totalThreads * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_subresultList, threadCountList, totalThreads * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_multresultList, threadCountList, totalThreads * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_modresultList, threadCountList, totalThreads * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_randNumList, randNumList, totalThreads * sizeof(int), hipMemcpyHostToDevice);
	
	// Execute each operation  
	addCUDA<<<numBlocks,totalThreads>>> (dev_addresultList, dev_randNumList);
	subCUDA<<<numBlocks,totalThreads>>> (dev_subresultList, dev_randNumList);
	multCUDA<<<numBlocks,totalThreads>>> (dev_multresultList, dev_randNumList);
	modCUDA<<<numBlocks,totalThreads>>> (dev_modresultList, dev_randNumList);

	// Bring result from device to host
	hipMemcpy(addresultList, dev_addresultList, totalThreads * sizeof(int), hipMemcpyDeviceToHost); 
	hipMemcpy(subresultList, dev_subresultList, totalThreads * sizeof(int), hipMemcpyDeviceToHost); 
	hipMemcpy(multresultList, dev_multresultList, totalThreads * sizeof(int), hipMemcpyDeviceToHost); 
	hipMemcpy(modresultList, dev_modresultList, totalThreads * sizeof(int), hipMemcpyDeviceToHost); 

	// Turned of to minimize printing
	printArray("Add Result", addresultList, totalThreads);
	printArray("Sub Result", subresultList, totalThreads);
	printArray("Mult Result", multresultList, totalThreads);
	printArray("Mod Result", modresultList, totalThreads);
	
	// Free reserved memory
	hipFree(dev_addresultList);
	hipFree(dev_subresultList);
	hipFree(dev_multresultList);
	hipFree(dev_modresultList);
	hipFree(dev_randNumList);
}

void timeTest(int numBlocks, int totalThreads) {
	// Set up variables for timing
	clock_t start, end;
	double timePassedMiliSeconds;

	// Set up paged memory space 
	int* threadCountList = (int*) malloc(totalThreads * sizeof(int));
	int* randNumList = (int*) malloc(totalThreads * sizeof(int));

	// Populate paged memory arrays
	for ( int idx = 0; idx < totalThreads; idx++ ) {
    	threadCountList[idx] = idx; 
		randNumList[idx] = rand() % 4;
   	}
	
	// Show generated values
	// Turned of to minimize printing
	printArray("Thread Count List", threadCountList, totalThreads);
	printArray("Random Number List", randNumList, totalThreads);
	
	// Run and time operations using register memory
	start = clock();
	runOperations(numBlocks, totalThreads, threadCountList, randNumList);
	end = clock();
	timePassedMiliSeconds = (double) (end - start) * 1000 / CLOCKS_PER_SEC;
	printf("\nRegister Memory Time: %f Miliseconds\n", timePassedMiliSeconds);
}
int main(int argc, char** argv) {
	// Based on the work of Andrew Krepps
	
	// Set default values in case arguments don't come in command line.
	int totalThreads = 1024;
	int blockSize = 256;

	// read command line arguments
	if (argc >= 2) {
		totalThreads = atoi(argv[1]);
	}
	if (argc >= 3) {
		blockSize = atoi(argv[2]);
	}

	int numBlocks = totalThreads/blockSize;

	// validate command line arguments
	if (totalThreads % blockSize != 0) {
		++numBlocks;
		totalThreads = numBlocks*blockSize;
		
		printf("Warning: Total thread count is not evenly divisible by the block size\n");
		printf("The total number of threads will be rounded up to %d\n", totalThreads);
	}

	printf("Total Threads: %d\nBlock Size: %d\n", totalThreads, blockSize);
	
	timeTest(numBlocks, totalThreads);

	return 0;
}
