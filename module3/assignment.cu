//Based on the work of Andrew Krepps
#include <stdio.h>

struct ThreadVariables {
	int threadCountList[];
 	int randNumList[];
	int resultList[];
};

int main(int argc, char** argv)
{
	// read command line arguments
	int totalThreads = 64;
	int blockSize = 1;
	
	if (argc >= 2) {
		totalThreads = atoi(argv[1]);
	}
	if (argc >= 3) {
		blockSize = atoi(argv[2]);
	}

	int numBlocks = totalThreads/blockSize;

	// validate command line arguments
	if (totalThreads % blockSize != 0) {
		++numBlocks;
		totalThreads = numBlocks*blockSize;
		
		printf("Warning: Total thread count is not evenly divisible by the block size\n");
		printf("The total number of threads will be rounded up to %d\n", totalThreads);
	}

	int arraySize = totalThreads;

	// Initialize operation arrays
	pthread_t threads[arraySize];
	int threadCountList[arraySize];
 	int randNumList[arraySize];
	int resultList[arraySize];

	struct ThreadVariables *cpuVar;
    cpuVar = malloc(sizeof(struct threadVariables));

	(*cpuVar).threadCountList = threadCountList;
	(*cpuVar).randNumList = randNumList;
	(*cpuVar).resultList = resultList;

   	// Populate elements of both arrays          
   	for ( int idx = 0; idx < arraySize; idx++ ) {
    	threadCountList[idx] = idx; 
		randNumList[idx] = rand() % 4;
   	}

	// Test using gpu threads
	int *dev_threadCountList, *dev_randNumList, *dev_c;
	
	hipMalloc((void**)&dev_threadCountList, arraySize * sizeof(int));
	
	hipMalloc((void**)&dev_randNumList, arraySize * sizeof(int));
	
	hipMalloc((void**)&dev_resultList, arraySize * sizeof(int));

	hipMemcpy(dev_threadCountList, threadCountList, arraySize * sizeof(int), hipMemcpyHostToDevice);
	
	hipMemcpy(dev_randNumList, randNumList, arraySize * sizeof(int), hipMemcpyHostToDevice);

	auto start = std::chrono::high_resolution_clock::now();
	
	addCUDA<<<numBlocks,blockSize>>> (dev_threadCountList, dev_randNumList, dev_resultList);
	subCUDA<<<numBlocks,blockSize>>> (dev_threadCountList, dev_randNumList, dev_resultList);
	multCUDA<<<numBlocks,blockSize>>> (dev_threadCountList, dev_randNumList, dev_resultList);
	modCUDA<<<numBlocks,blockSize>>> (dev_threadCountList, dev_randNumList, dev_resultList);
	
	auto stop = std::chrono::high_resolution_clock::now();

	hipFree(dev_threadCountList);
	
	hipFree(dev_randNumList);
	
	hipFree(dev_resultList);

	std::cout <endl<< " Time elapsed on GPU: " << std::chrono::duration_castchrono::nanoseconds>(stop - start).count() << "ns\n";

	// Test using cpu threads
	auto startCpu = std::chrono::high_resolution_clock::now();


	for(int idx = 0; idx < arraySize ; idx++) {
		thread = pthread_create(&threads[idx], NULL, cpuMain, (void *)cpuVar)
		if (thread) {
			printf("Error:unable to create thread, %d\n", thread);
			exit(-1);
		}
	}

	auto stopCpu = std::chrono::high_resolution_clock::now();

	std::cout << " Time elapsed on CPU: " << std::chrono::duration_castchrono::nanoseconds>(stopCpu - startCpu).count() << "ns\n";

	return 0;
}

void cpuMain(void* threadVariables) {
	
	// Initialize operation arrays
	struct ThreadVariables *cpuVariables = (struct ThreadVariables*)threadVariables;
	thread_idx = pthread_getthreadid_np();

	add((*cpuVariables).threadCountList, (*cpuVariables).randNumList , (*cpuVariables).resultList , (int) thread_idx);
	sub((*cpuVariables).threadCountList, (*cpuVariables).randNumList , (*cpuVariables).resultList , (int) thread_idx);
	mult((*cpuVariables).threadCountList, (*cpuVariables).randNumList , (*cpuVariables).resultList , (int) thread_idx);
	mod((*cpuVariables).threadCountList, (*cpuVariables).randNumList , (*cpuVariables).resultList , (int) thread_idx);
}

void addCUDA(int *threadCountList, int *randNumList, int *resultList) { 
	int idx = threadIdx.x + blockIdx.x * blockDim.x; 
	resultList[idx] = threadCountList[idx] + randNumList[idx]; 
}

void subCUDA(int *threadCountList, int *randNumList, int *resultList) { 
	int idx = threadIdx.x + blockIdx.x * blockDim.x; 
	resultList[idx] = threadCountList[idx] - randNumList[idx]; 
}

void multCUDA(int *threadCountList, int *randNumList, int *resultList) { 
	int idx = threadIdx.x + blockIdx.x * blockDim.x; 
	resultList[idx] = threadCountList[idx] * randNumList[idx]; 
}

void modCUDA(int *threadCountList, int *randNumList, int *resultList) { 
	int idx = threadIdx.x + blockIdx.x * blockDim.x; 
	resultList[idx] = threadCountList[idx] % randNumList[idx]; 
}

void add(int *threadCountList, int *randNumList, int *resultList, int thread_idx) { 
	resultList[thread_idx] = threadCountList[thread_idx] + randNumList[thread_idx]; 
}

void sub(int *threadCountList, int *randNumList, int *resultList, int thread_idx) { 
	resultList[thread_idx] = threadCountList[thread_idx] - randNumList[thread_idx]; 
}

void mult(int *threadCountList, int *randNumList, int *resultList, int thread_idx) { 
	resultList[thread_idx] = threadCountList[thread_idx] * randNumList[thread_idx]; 
}

void mod(int *threadCountList, int *randNumList, int *resultList, int thread_idx) { 
	resultList[thread_idx] = threadCountList[thread_idx] % randNumList[thread_idx]; 
}
